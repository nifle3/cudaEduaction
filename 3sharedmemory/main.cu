#include <iostream>

template <typename T>
class CudaMemory 
{
private:
    T* pointer;
    size_t memSize;

    void memCpy(T* dst, const T* src, hipMemcpyKind kind)
    {
        auto result = hipMemcpy(dst, src, this->memSize, kind)
        if (result != hipSuccess)
        {
            throw "Cuda memcpy error!";
        } 
    }
public:
    CudeMemory(size_t mem) : memSize(mem) 
    {
        auto result = hipMalloc(&this->pointer, mem);
        if (result != hipSuccess)
        {
            throw "Cuda malloc error!";
        }
    }

    ~CudaMemory()
    {
        hipFree(this->pointer);
    }

    T* getPointer() { return this->pointer; }

    void memCpyToHost(T* dst)
    {
        this->memCpy(dst, this->pointer, hipMemcpyDeviceToHost);
    }

    void memCpyToDevice(const T* src)
    {
        this->memCpy(this->pointer, src, hipMemcpyHostToDevice);
    }
};

template <typename T, typename = typename std::enable_if_t<std::is_arithmetic_v<T>>>
void testCase()
{

}

int main() 
{
    testCase<int>();
    testCase<float>();
    testCase<double>();
    testCase<long long>();
    
    return 0;
}